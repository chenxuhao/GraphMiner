#include "hip/hip_runtime.h"
// Copyright (c) 2020 MIT
// Author: Xuhao Chen
#include <hipcub/hipcub.hpp>
#include "graph_gpu.h"
#include "pattern.hh"
#include "operations.cuh"
#include "hip/hip_runtime_api.h"
#include "cuda_launch_config.hpp"

#define BLK_SZ BLOCK_SIZE
typedef hipcub::BlockReduce<AccType, BLK_SZ> BlockReduce;
#if DO_COUNT
#include "diamond_count.cuh"
#else
#include "diamond_nested.cuh"
#endif
#include "rectangle_nested.cuh"
#include "house_edge_warp_nested.cuh"
#include "pentagon_edge_warp_nested.cuh"

void SglSolver(Graph &g, Pattern &p, uint64_t &total, int, int) {
  size_t memsize = print_device_info(0);
  vidType nv = g.num_vertices();
  eidType ne = g.num_edges();
  auto md = g.get_max_degree();
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";
  if (memsize < mem_graph) std::cout << "Graph too large. Unified Memory (UM) required\n";

  GraphGPU gg(g);
  AccType h_total = 0, *d_total;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_total, sizeof(AccType)));
  CUDA_SAFE_CALL(hipMemcpy(d_total, &h_total, sizeof(AccType), hipMemcpyHostToDevice));
  int k = 4;
  if (p.is_house() || p.is_pentagon()) k = 5;

  size_t nthreads = BLK_SZ;
  size_t nwarps = BLK_SZ/WARP_SIZE;
  size_t ntasks = nv;
  ntasks = gg.init_edgelist(g, 1);
  std::cout << "Edge parallel: edgelist size = " << ntasks << "\n";
  size_t nblocks = (ntasks-1)/nwarps+1;
  if (nblocks > 65536) nblocks = 65536;
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  int max_blocks_per_SM;
  if (p.is_house()) {
    max_blocks_per_SM = maximum_residency(house_warp_edge_nested, nthreads, 0);
  } else if (p.is_pentagon()) {
    max_blocks_per_SM = maximum_residency(pentagon_warp_edge_nested, nthreads, 0);
  } else if (p.is_rectangle()) {
    max_blocks_per_SM = maximum_residency(rectangle_warp_edge_nested, nthreads, 0);
  } else {
#if DO_COUNT
    max_blocks_per_SM = maximum_residency(diamond_warp_edge_count, nthreads, 0);
#else
    max_blocks_per_SM = maximum_residency(diamond_warp_edge_nested, nthreads, 0);
#endif
  }
  size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  if (p.is_diamond() || p.is_pentagon()) nblocks = std::min(3*max_blocks, nblocks);
  std::cout << p.get_name() << " max_blocks_per_SM = " << max_blocks_per_SM << "\n";

  vidType *frontier_list; // each warp has (k-3) vertex sets; each set has size of max_degree
#ifndef DO_COUNT
  size_t per_block_vlist_size = nwarps * size_t(k-3) * size_t(md) * sizeof(vidType);
  if (k > 3) {
    auto nb = int64_t(memsize*0.95 - mem_graph) / int64_t(per_block_vlist_size);
    if (nb < nblocks) nblocks = nb;
  }
  size_t list_size = nblocks * per_block_vlist_size;
  if (p.is_rectangle()) list_size = 0;
  std::cout << "frontier list size " << list_size/(1024*1024) << " MB\n";
  if (list_size > 0) CUDA_SAFE_CALL(hipMalloc((void **)&frontier_list, list_size));
#endif
  std::cout << "CUDA subgraph listing (" << nblocks << " CTAs, " << nthreads << " threads/CTA) ...\n";

  Timer t;
  t.Start();
  hipProfilerStart();
  if (p.is_house()) {
    house_warp_edge_nested<<<nblocks, nthreads>>>(ntasks, gg, frontier_list, md, d_total);
  } else if (p.is_pentagon()) {
    pentagon_warp_edge_nested<<<nblocks, nthreads>>>(ntasks, gg, d_total);
  } else if (p.is_rectangle()) {
    rectangle_warp_edge_nested<<<nblocks, nthreads>>>(ntasks, gg, d_total);
    //rectangle_warp_edge_nested_balanced<<<nblocks, nthreads>>>(ntasks, gg, d_total);
  } else {
#if DO_COUNT
    diamond_warp_edge_count<<<nblocks, nthreads>>>(ntasks, gg, frontier_list, md, d_total);
#else
    diamond_warp_edge_nested<<<nblocks, nthreads>>>(ntasks, gg, frontier_list, md, d_total);
#endif
  }
  hipProfilerStop();
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();

  std::cout << "runtime [cuda_base] = " << t.Seconds() << " sec\n";
  CUDA_SAFE_CALL(hipMemcpy(&h_total, d_total, sizeof(AccType), hipMemcpyDeviceToHost));
  total = h_total;
  CUDA_SAFE_CALL(hipFree(d_total));
}

